#include "hip/hip_runtime.h"
#include <Universe/Camera.cuh>
#include <iostream>

using namespace std;

__global__ void render(DeviceCamera **d_camera_ptr,FrameBuffer **d_fb_ptr) {

	DeviceCamera* d_camera = *d_camera_ptr;
	FrameBuffer* d_fb = *d_fb_ptr;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= d_camera -> d_width) || (j >= d_camera -> d_height)) return;

	int pixel_index = j*(d_camera -> d_width) + i;
}


__global__ void CameraInit(
	int width,
	int height,
	DeviceCamera **d_camera_ptr,
	FrameBuffer **d_fb_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			//printf("CameraInitBegin\n");
			*d_camera_ptr = new DeviceCamera(width,height);
			*d_fb_ptr = new FrameBuffer(width*height);

		}
}
__global__ void CameraCleanup(
	DeviceCamera **d_camera_ptr,
	FrameBuffer **d_fb_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			delete *d_camera_ptr;
			delete *d_fb_ptr;
		}
}

__device__ DeviceCamera::DeviceCamera(int width,int height){
	d_width = width;
	d_height = height;
}
__device__ DeviceCamera::~DeviceCamera(){
}
__device__ FrameBuffer::FrameBuffer(int s){
	size = s;
	d_frame_buffer = new vec3[size];
}
__device__ FrameBuffer::~FrameBuffer(){
	delete d_frame_buffer;
}

Camera::Camera(int width,int height) {
	h_width = width;
	h_height = height;

	h_image.resize(width*height,vec3(0.0f,0.0f,0.0f)); 

	checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(DeviceCamera *)));
	checkCudaErrors(hipMalloc((void **)&d_fb, sizeof(FrameBuffer *)));

	CameraInit<<<1,1>>>(
		h_width,
		h_height,
		d_camera,
		d_fb);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

Camera::~Camera() {
	CameraCleanup<<<1,1>>>(
		d_camera,
		d_fb);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(d_camera));
	checkCudaErrors(hipFree(d_fb));
}

const std::vector<vec3>& Camera::getImage(){

	render<<<1,1>>>(d_camera,d_fb);

	return h_image;
}