#include "hip/hip_runtime.h"
#include <Universe/Camera.cuh>
#include <iostream>

using namespace std;

__global__ void Render(DeviceCamera **d_camera_ptr,vec3 *d_fb) {

	DeviceCamera* d_camera = *d_camera_ptr;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= d_camera -> d_width) || (j >= d_camera -> d_height)) return;

	int pixel_index = j*(d_camera -> d_width) + i;

	d_fb[pixel_index] = vec3((float)i/(d_camera -> d_width),(float)j / (d_camera -> d_height),1.0f);
}


__global__ void CameraInit(
	int width,
	int height,
	DeviceCamera **d_camera_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			//printf("CameraInitBegin\n");
			*d_camera_ptr = new DeviceCamera(width,height);
		}
}
__global__ void CameraCleanup(
	DeviceCamera **d_camera_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			delete *d_camera_ptr;
		}
}

__device__ DeviceCamera::DeviceCamera(int width,int height){
	d_width = width;
	d_height = height;
}
__device__ DeviceCamera::~DeviceCamera(){
}
FrameBuffer::FrameBuffer(int s){
	size = s;
	checkCudaErrors(hipMalloc((void **)&d_frame_buffer, sizeof(vec3)*size));

	h_frame_buffer.resize(size,vec3(0.0f,0.0f,0.0f)); 
}

FrameBuffer::~FrameBuffer(){
	checkCudaErrors(hipFree(d_frame_buffer));
}
void FrameBuffer::copyDeviceToHost(){
	checkCudaErrors(hipMemcpy(h_frame_buffer.data(),d_frame_buffer,size*sizeof(vec3),hipMemcpyDeviceToHost));
}

Camera::Camera(int width,int height) :
	fb(width*height){

	/* Max threads per block in my pc, divided equaly across the width and height */
	const unsigned int tx = 32;
	const unsigned int ty = 32;
	
	blocks = dim3(width/tx+1,height/ty+1);
	threads = dim3(tx,ty);

	h_width = width;
	h_height = height;

	checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(DeviceCamera *)));

	CameraInit<<<1,1>>>(
		h_width,
		h_height,
		d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

Camera::~Camera() {
	CameraCleanup<<<1,1>>>(
		d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(d_camera));
}

const std::vector<vec3>& Camera::getImage(){
	
	Render<<<blocks,threads>>>(d_camera,fb.d_frame_buffer);
	fb.copyDeviceToHost();

	return fb.h_frame_buffer;
}