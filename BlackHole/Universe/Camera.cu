#include "hip/hip_runtime.h"
#include <Universe/Camera.cuh>
#include <Universe/FrameBuffer.cuh>
#include <Universe/Renderer.cuh>

#include <iostream>

using namespace std;

__global__ void CameraInit(
	int width,
	int height,
	DeviceCamera **d_camera_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			*d_camera_ptr = new DeviceCamera(width,height);
		}
}
__global__ void CameraCleanup(
	DeviceCamera **d_camera_ptr) {
		if (threadIdx.x == 0 && blockIdx.x == 0) {
			delete *d_camera_ptr;
		}
}

__device__ DeviceCamera::DeviceCamera(int width,int height){
	d_width = width;
	d_height = height;
}
__device__ DeviceCamera::~DeviceCamera(){
}

Camera::Camera(int width,int height) :
	fb(width*height){

	/* Max threads per block in my pc, divided equaly across the width and height */
	const unsigned int tx = 32;
	const unsigned int ty = 32;
	
	blocks = dim3(width/tx+1,height/ty+1);
	threads = dim3(tx,ty);

	h_width = width;
	h_height = height;

	checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(DeviceCamera *)));

	CameraInit<<<1,1>>>(
		h_width,
		h_height,
		d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

Camera::~Camera() {
	CameraCleanup<<<1,1>>>(
		d_camera);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(d_camera));
}

const std::vector<vec3>& Camera::getImage(UniformsList l){
	
	Render<<<blocks,threads>>>(d_camera,fb.d_frame_buffer,l);
	fb.copyDeviceToHost();

	return fb.h_frame_buffer;
}