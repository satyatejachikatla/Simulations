#include "hip/hip_runtime.h"
#include <Universe/Renderer.cuh>

__global__ void Render(DeviceCamera **d_camera_ptr,vec3 *d_fb,UniformsList l) {

	DeviceCamera* d_camera = *d_camera_ptr;

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= d_camera -> d_width) || (j >= d_camera -> d_height)) return;

	int pixel_index = j*(d_camera -> d_width) + i;

	float s,c;
	float t = *l.u_time *1;
	sincos(t,&s,&c);

	d_fb[pixel_index] = vec3(s,1-s,1.0f);
}


