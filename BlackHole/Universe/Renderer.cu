#include "hip/hip_runtime.h"
#include <CudaVector.cuh>
#include <Universe/Renderer.cuh>


#define PI 3.1415926535897932384626433832795

#define MAX_DIST        100.
#define MAX_STEPS       100
#define SURF_DIST       .01

__device__ float clamp(float y , float a, float b ) {
	return (y > b ? b : y) < a ? a  : y ;
}

__device__ float GetDist(vec3 p){
	vec4 s = vec4(0.f,0.f,0.f,1.f);

 	float sphereDist = (p - vec3(s.x(),s.y(),s.z())).length() - s.w();
 	float planeDist = p.y();

 	float d = fminf(sphereDist,planeDist);

 	return d;

}

__device__ float RayMarch(vec3 ro,vec3 rd){
	float dO = 0;

	for(int i=0;i<MAX_STEPS;i++){
		vec3 p = ro+rd*vec3(dO,dO,dO);
		float dS = GetDist(p);
		dO += dS;
		if(dO > MAX_DIST || dS < SURF_DIST) break;
	}

	return dO;
} 

__device__ vec3 GetNormal(vec3 p){
	float d = GetDist(p);
	vec2 e = vec2(0.01f,0.f);

	vec3 n = vec3(d,d,d)- vec3(
				GetDist(p-vec3(e.x(),e.y(),e.y())),
				GetDist(p-vec3(e.y(),e.x(),e.y())),
				GetDist(p-vec3(e.y(),e.y(),e.x()))
			);

	return unit_vector(n);
}

__device__ float GetLight(vec3 p){
	vec3 lightPosition = vec3(0.f,5.f,6.f);

	vec3 l = unit_vector(lightPosition-p);
	vec3 n = GetNormal(p);

	float diff = clamp(dot(n,l),0.f,1.f);

	float d = RayMarch(p+n*SURF_DIST*1.2f,l);
	if (d<(lightPosition-p).length()) diff *= .1f;

	return diff;
}

__global__ void Render(DeviceCamera **d_camera_ptr,vec3 *d_fb,UniformsList l) {

	DeviceCamera* d_camera = *d_camera_ptr;

	float i = threadIdx.x + blockIdx.x * blockDim.x;
	float j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= d_camera -> d_width) || (j >= d_camera -> d_height)) return;

	int pixel_index = j*(d_camera -> d_width) + i;

	vec2 fragCoords = vec2(i,j);
	vec2 uv = (fragCoords-.5*(*l.u_resolution))/(l.u_resolution->y());

	/*camera start*/
	float zoom = 1.;
	vec3 ro = vec3(2.,1.,2.);
	vec3 lookat = vec3(0.,0.,0.);
	vec3 u_world = vec3(0.,1.,0.);
	vec3 f = unit_vector(lookat-ro);
	vec3 r = cross(u_world,f);
	vec3 u = cross(f,r);

	vec3 c = ro + f*zoom;
	vec3 k = c + uv.x()*r + uv.y()*u;

	vec3 rd = unit_vector(k-ro);
	/*camera end*/


	float d = RayMarch(ro,rd);

	vec3 p = ro+rd*d;

	float diff = GetLight(p);
	d_fb[pixel_index] = vec3(diff,diff,diff);

}


