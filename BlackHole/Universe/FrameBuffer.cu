#include <Universe/FrameBuffer.cuh>
#include <CudaErrors.cuh>

FrameBuffer::FrameBuffer(int s){
	size = s;
	checkCudaErrors(hipMalloc((void **)&d_frame_buffer, sizeof(vec3)*size));

	h_frame_buffer.resize(size,vec3(0.0f,0.0f,0.0f)); 
}

FrameBuffer::~FrameBuffer(){
	checkCudaErrors(hipFree(d_frame_buffer));
}

void FrameBuffer::copyDeviceToHost(){
	checkCudaErrors(hipMemcpy(h_frame_buffer.data(),d_frame_buffer,size*sizeof(vec3),hipMemcpyDeviceToHost));
}
