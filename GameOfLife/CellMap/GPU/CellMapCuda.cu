#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <CellMap/GPU/CellMapCuda.cuh>
#include <CudaErrors.cuh>
#include <cmath>

using namespace std;

#define MAX_THREADS_IN_AXIS 32.0

__host__ __device__ inline int mod(int value,int m) {
	int modulo = value % m;
	if (modulo < 0) modulo += m;

	return modulo;
}

#define IDX(a,b)  mod((a),width) + mod((b),height) * width

__device__ bool IsInbounds(int x,int y,int width,int height,bool edge_wrap) {
	if(edge_wrap) return true;
	return (0 <= x  && x < width && 0 <= y && y < height);
}

__device__ int CellMapCudaGetNeighbourCount(bool* cells_gpu,int x,int y,int width,int height,bool edge_wrap) {

	int count=0;

	if ( IsInbounds(x-1,y-1,width,height,edge_wrap) ) count += cells_gpu[IDX(x-1,y-1)];
	if ( IsInbounds(x-1,y,width,height,edge_wrap)   ) count += cells_gpu[IDX(x-1,y)];
	if ( IsInbounds(x-1,y+1,width,height,edge_wrap) ) count += cells_gpu[IDX(x-1,y+1)];
	if ( IsInbounds(x,y-1,width,height,edge_wrap)   ) count += cells_gpu[IDX(x,y-1)];
	if ( IsInbounds(x,y+1,width,height,edge_wrap)   ) count += cells_gpu[IDX(x,y+1)];
	if ( IsInbounds(x+1,y-1,width,height,edge_wrap) ) count += cells_gpu[IDX(x+1,y-1)];
	if ( IsInbounds(x+1,y,width,height,edge_wrap)   ) count += cells_gpu[IDX(x+1,y)];
	if ( IsInbounds(x+1,y+1,width,height,edge_wrap) ) count += cells_gpu[IDX(x+1,y+1)];

	return count;
}


__global__ void CellMapCudaStep(bool* cells_gpu,bool* cells_gpu_temp,int width,int height,bool edge_wrap) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;

	if(i >= width || j >= height) return;

	int index = IDX(i,j);

	int neighbours_alive_count = CellMapCudaGetNeighbourCount(cells_gpu,i,j,width,height,edge_wrap);

	switch(neighbours_alive_count) {
		case 0 :
		case 1 : cells_gpu_temp[index] = false; break;
		case 2 : cells_gpu_temp[index] = cells_gpu[index]; break;
		case 3 : cells_gpu_temp[index] = true; break;
		case 4 :
		case 5 : 
		case 6 :
		case 7 :
		case 8 : cells_gpu_temp[index] = false; break;
	}

}

CellMapCuda::CellMapCuda(int width, int height, bool edge_wrap) :
	CellMap(width,height,edge_wrap) {
		checkCudaErrors(hipMallocManaged((void **)&cells_gpu, height*width*sizeof(*cells_gpu)));
		checkCudaErrors(hipMallocManaged((void **)&cells_gpu_temp, height*width*sizeof(*cells_gpu_temp)));

		checkCudaErrors(hipMemset(cells_gpu,0,height*width*sizeof(*cells_gpu)));
		checkCudaErrors(hipMemset(cells_gpu_temp,0,height*width*sizeof(*cells_gpu_temp)));
}
CellMapCuda::~CellMapCuda(){
	checkCudaErrors(hipFree(cells_gpu));
	checkCudaErrors(hipFree(cells_gpu_temp));
}


void CellMapCuda::Step(int step_count) {

	dim3 blocks(ceil(width/MAX_THREADS_IN_AXIS),ceil(height/MAX_THREADS_IN_AXIS));
	dim3 threads(MAX_THREADS_IN_AXIS,MAX_THREADS_IN_AXIS);

	bool* t;

	for(int c=0;c<step_count;c++) {
		CellMapCudaStep<<<blocks,threads>>>(cells_gpu,cells_gpu_temp,width,height,edgeWrap);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());
		t = cells_gpu;
		cells_gpu = cells_gpu_temp;
		cells_gpu_temp = t;
	}

}

void CellMapCuda::MakeCellAlive(int x, int y) {
	CellMap::MakeCellAlive(x,y);
	cells_gpu[IDX(x,y)] = true;
}

void CellMapCuda::MakeCellDie(int x, int y) {
	CellMap::MakeCellDie(x,y);
	cells_gpu[IDX(x,y)] = false;
}

bool CellMapCuda::GetCell(int x, int y){
	(*cells)[IDX(x,y)] = cells_gpu[IDX(x,y)];
	return (*cells)[IDX(x,y)];
}